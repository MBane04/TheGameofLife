
#include <hip/hip_runtime.h>
#include <stdio.h>

#define WIDTH 26 //201x201 after 
#define HEIGHT 26

#define BACKGROUND '-'


char grid[HEIGHT][WIDTH];

void gridSetup()
{
    for(size_t i = 0; i < HEIGHT; i++)
    
        for(size_t j = 0; j < WIDTH; j++)
        {
            grid[i][j] = '-';
        }
     
}

void gridPrint()
{
    for(size_t i = 0; i < HEIGHT; i++)
    {
        for(size_t j = 0; j < WIDTH; j++)
        {
            printf("%c", grid[i][j]);
        }
        printf("\n");
    }
}

int main()
{
    gridSetup();
    gridPrint();
    return 0;
}