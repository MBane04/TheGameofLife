//https://www.youtube.com/watch?v=I0ndY1Y4avQ&list=PLRnI_2_ZWhtCcyinpkAbk5M2A4UhdOP3A&index=1
//4:45


#include <hip/hip_runtime.h>
#include <stdio.h>

#define WIDTH 26 //201x201 after 
#define HEIGHT 26

#define BACKGROUND '-'
#define CELL '#'


char grid[HEIGHT][WIDTH];

void gridSetup()
{
    for(size_t i = 0; i < HEIGHT; i++)
    
        for(size_t j = 0; j < WIDTH; j++)
        {
            grid[i][j] = 'BACKGROUND';
        }
     
}

void gridPrint()
{
    for(size_t i = 0; i < HEIGHT; i++)
    {
        for(size_t j = 0; j < WIDTH; j++)
        {
            printf("%c", grid[i][j]);
        }
        printf("\n");
    }
}

int main()
{
    gridSetup();
    grid[HEIGHT/2][WIDTH/2] = CELL;  
    gridPrint();
    return 0;
}